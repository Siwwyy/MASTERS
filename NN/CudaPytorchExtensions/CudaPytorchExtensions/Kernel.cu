﻿

#include "Kernel.cuh"



void Cuda_Kernel::kernel_double(double* A, double* B, double* C, const std::size_t& array_size)
{
	//kernel<double>(A, B, C, array_size);

    double* d_A, * d_B, * d_C;
    unsigned int* d_ArraySize;


    hipMalloc((void**)&d_A        , array_size * sizeof(double));
    hipMalloc((void**)&d_B        , array_size * sizeof(double));
    hipMalloc((void**)&d_C        , array_size * sizeof(double));
    hipMalloc((void**)&d_ArraySize, sizeof(unsigned int));


    hipMemcpy(d_A        , A, array_size * sizeof(double), HostToDevice);
    hipMemcpy(d_B        , B, array_size * sizeof(double), HostToDevice);
    hipMemcpy(d_ArraySize, &array_size, sizeof(unsigned int), HostToDevice);


    dim3 blockSize(512, 1, 1);
    dim3 gridSize(512 / array_size + 1, 1);

    vector_addition_kernel<double> <<<10, 10, 10>>> (d_A, d_B, d_C, d_ArraySize);


    hipMemcpy(C, d_C, array_size * sizeof(double), DeviceToHost);

    //Free allocated memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_ArraySize);
}

//template<typename Runnable>
//void Cuda_Kernel::test_function(const Runnable& runnable)
//{
//	
//}